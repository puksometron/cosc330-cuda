#include "hip/hip_runtime.h"
/*
 * COSC330 Assignment 04 - By Vladimir Ovechkin
 * This program generates a mandelbrot using CUDA.
 *
 * To compile this program, use the "make program" command from the Terminal.
 * To run this program, enter "make run".
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "bmpfile.h"

#define WIDTH 1920
#define HEIGHT 1080
#define MAX_ITER 1000

#define COLOUR_DEPTH 255
#define COLOUR_MAX 240.0
#define GRADIENT_COLOUR_MAX 230.0

/*
 * Performs colour mixing on the value of x.
 * color -> The RGB colour values.
 * x-> The input value for determining the position on the gradient.
 * min -> The minimum value any colour can take during mixing.
 * max -> The maximum value any colour can take during mixing.
 */
void GroundColorMix(double* color, double x, double min, double max) 
{
    // Calculate positive and negative slopes for colour component interpolation
    double posSlope = (max - min) / 60;
    double negSlope = (min - max) / 60;

    // Determine the appropriate colour mixing on the value of x
    if(x < 60)
    {
        color[0] = max;
        color[1] = posSlope * x + min;
        color[2] = min;
        return;
    }

    else if (x < 120)
    {
        color[0] = negSlope * x + 2.0 * max + min;
        color[1] = max;
        color[2] = min;
        return;
    }

    else if (x < 180)
    {
        color[0] = min;
        color[1] = max;
        color[2] = posSlope * x - 2.0 * max + min;
        return;
    }

    else if (x < 240)
    {
        color[0] = min;
        color[1] = negSlope * x + 4.0 * max + min;
        color[2] = max;
        return;
    }

    else if (x < 300)
    {
        color[0] = posSlope * x - 4.0 * max + min;
        color[1] = min;
        color[2] = max;
        return;
    }

    else
    {
        color[0] = max;
        color[1] = min;
        color[2] = negSlope * x + 6 * max;
        return;
    }
}

/*
 * Produce a mandelbrot.
 * d_result ->  Pointer to the memory where the mandelbrot will be stored.
 * width -> Output BMP width.
 * height -> Output BMP height.
 * xCenter -> The x-coordinate for where the mandelbrot is generated.
 * yCenter -> The y-coordinate for where the mandelbrot is generated.
 * resolution -> The level of detail in the mandelbrot.
 */
__global__ void mandelbrotKernel(float *d_result, int width, int height, float xCenter, float yCenter, float resolution) 
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) 
    {
        // Calculate coordinates for this pixel
        float x = xCenter + (col - width / 2.0f) / resolution;
        float y = yCenter + (height / 2.0f - row) / resolution;

        // Mandelbrot calculations
        float a = 0.0f, b = 0.0f;
        int iter = 0;
        while (iter < MAX_ITER && a * a + b * b <= 4.0f) 
        {
            float aTemp = a * a - b * b + x;
            b = 2.0f * a * b + y;
            a = aTemp;
            ++iter;
        }

        // Map iter to color and store in d_result
        d_result[row * width + col] = (float)iter;
    }
}

int main() 
{
    // Allocate host memory for the result
    float *h_result = (float *)malloc(WIDTH * HEIGHT * sizeof(float));

    // Allocate device memory for the result
    float *d_result;
    hipMalloc((void **)&d_result, WIDTH * HEIGHT * sizeof(float));

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
    mandelbrotKernel<<<numBlocks, threadsPerBlock>>>(d_result, WIDTH, HEIGHT, -0.55f, 0.6f, 8700.0f);

    // Copy the result from device to host
    hipMemcpy(h_result, d_result, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);

    // Generate and save the bitmap image
    bmpfile_t *bmp = bmp_create(WIDTH, HEIGHT, 32);
    for (int col = 0; col < WIDTH; col++) 
    {
        for (int row = 0; row < HEIGHT; row++) 
        {
            float normalizedValue = h_result[row * WIDTH + col] / MAX_ITER;
            double color[3];
            GroundColorMix(color, COLOUR_MAX - normalizedValue * GRADIENT_COLOUR_MAX, 1, COLOUR_DEPTH);
            rgb_pixel_t pixel = {(uint8_t)color[0], (uint8_t)color[1], (uint8_t)color[2], 0};
            bmp_set_pixel(bmp, col, row, pixel);
        }
    }
    bmp_save(bmp, "mandelbrot.bmp");
    bmp_destroy(bmp);

    // Free memory
    free(h_result);
    hipFree(d_result);

    return 0;
}